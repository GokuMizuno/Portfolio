#include "hip/hip_runtime.h"
//Gordon Stangler
//Mandelbrot set via CUDA/OpenCV

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <opencs2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#define HEIGHT 512
#define WIDTH 512
#define MAX_ITER 1000

void MandelbrotGPU(char*);
__global__ void calc(char* image_buffer);

//#define cudeAssertSuccess(ans) { _cudaAssertSuccess((ans), __FILE__, __LINE__); }
inline void _cudaAssertSuccess(hipError_t code, char* file, int line)
{
	if (code !- hipSuccess)
	{
		fprintf(stderr, "cudaAssertSuccess: %s %s %d\n", cudaGetError(code), file, line);
		exit(code);
	}
}

int main(int argc, char** argv)
{
	IplImage* image_output = cvCreateImage(cvSize(WIDTH, HEIGHT), IPL_DEPTH_8U, 1);
	mandelbrotGPU(image_output->imageData);
	cvShowImage("GPU", image_output);
	waitKey(0);
	cvReleaseImage(&image_output);
	return 0;
}

void MandelbrotGPU(char* image_buffer)
{
	char* dImageBuffer;
	cudaAssertSuccess(hipMalloc(&dImageBuffer, HEIGHT*WIDTH, hipMemcpyDeviceToHost));
	cudaAssertSuccess(hipFree(DImageBuffer));
}

__global__ void calc(char* image_buffer)
{
	int row = blokcIdx.y * blockDim.y + threadIdx.y; //width
	int col = blockIdx.x * blockDim.x + threadIdx.x; //height
	int idx = row * WIDTH + col;

	if(col >= WIDTH || row >= HEIGHT) return;

	float x0 = ((float)col / WIDTH * 3.5f - 2.5f;
	float y0 = ((float)row / HEIGHT * 2.5f - 1.74f;
	float x = 0.0f;
	float y = 0.0f;
	int iter = 0;
	float xtemp;

	while((x*x - y*y <= 4.0f) && (iter < MAX_ITER))
	{
		xtemp = x*x - y*y + x0;
		y = 2.0f * x * y +y0;
		x = xtemp;
		iter++;
	}

	int color = 5 * iter;

	if (color >= 256) color = 0;
	image_buffer[idx] = color;
}
